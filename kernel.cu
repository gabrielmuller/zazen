#include "hip/hip_runtime.h"
#include "book/book.h"
#include "book/gpu_anim.h"
#include <cmath>

#define _ HANDLE_ERROR
const int DIM = 1024;

__global__ struct Voxel {
    /* Non-leaf voxel. */

    // 15 high bits: relative child pointer
    // 1 lowest bit: far flag TODO
    int16_t child;
    int8_t valid; // 8 flags of whether children are visible
    int8_t leaf;  // 8 flags of whether children are leaves
};

__global__ struct Leaf {
    /* Leaf voxel. */

    int8_t r, g, b, a;
};

__global__ struct Block {
    static const std::size_t element_size = 4;
    const size_t element_count;
    char* data = nullptr;
    char* front = nullptr;

    __device__ explicit Block(size_t element_count, char* data) :
            element_count(element_count),
            data(data) {
        front = data;
    }

    explicit Block(size_t element_count) : element_count(element_count) {
        data = new char[element_count * element_size];
        front = data;
    }


    ~Block() {
        delete[] data;
    }

    Block(Block&) = delete; // No copy constructor.

    Block& operator=(Block&) = delete; // No assigning.

    // Move constructor
    Block(Block&& rhs) :
            element_count(rhs.element_count),
            data(rhs.data) {
        rhs.data = nullptr;
    }

    // Move assignment operator
    Block& operator=(Block&& rhs) {
        if (this != &rhs) {
            operator delete(data);
            data = rhs.data;
            rhs.data = nullptr;
        }
        return *this;
    }

    template <class T>
    __device__ T& get(const std::size_t index) const {
        return ((T*) data)[index];
    }

    char* slot() {
        char* front_slot = front;
        front += element_size;
        return front_slot;
    }

    size_t size() {
        return element_count * element_size;
    }
};

__global__ void placeholder(uchar4 *ptr, int ticks) {
    // map from threadIdx/BlockIdx to pixel position
    const int pixel_x = threadIdx.x + blockIdx.x * blockDim.x;
    const int pixel_y = threadIdx.y + blockIdx.y * blockDim.y;
    const int offset = pixel_x + pixel_y * blockDim.x * gridDim.x;
    const float screen_x = pixel_x / (float) DIM;
    const float screen_y = pixel_y / (float) DIM;

    const float time = ticks * 0.01;

    ptr[offset].x = screen_x * (sin(time * 3) + 1) * 0.5 * 256;
    ptr[offset].y = (1 - screen_y) * (sin(time * 7) + 1) * 0.5 * 256;
    ptr[offset].z = (1 - screen_x) * screen_y * (cos(time * 13) + 1) * 128;
    ptr[offset].w = 0xff;
}

__global__ void kernel(char* data, size_t element_count) {
    Block* block = new Block(element_count, data);
    printf("Here goes nothing: '");
    printf("%c", block->get<Voxel>(block->get<Voxel>(0).child + 1).valid);
    printf("'\n");
}


void generate_frame(uchar4 *pixels, void*, int ticks) {
    dim3    grids(DIM/16, DIM/16);
    dim3    threads(16, 16);
    placeholder<<<grids,threads>>>(pixels, ticks);
}

int main(void) {
    Block block(6);
    Voxel* y = new (block.slot()) Voxel();
    y->child = 1;
    y->valid = 'U';
    new (block.slot()) Leaf();
    Voxel* z = new (block.slot()) Voxel();
    z->child = 2;
    z->valid = 'f';
    new (block.slot()) Leaf();
    new (block.slot()) Leaf();
    new (block.slot()) Leaf();

    char* dev_data;
    _( hipMalloc((void**) &dev_data, block.size()) );
    _( hipMemcpy(dev_data, block.data, block.size(), hipMemcpyHostToDevice) );


    kernel<<<1, 1>>>(dev_data, block.element_count);
    _( hipFree(dev_data) );
    GPUAnimBitmap  bitmap(DIM, DIM, NULL);
    bitmap.anim_and_exit((void(*)(uchar4*,void*,int)) generate_frame, NULL);
}
